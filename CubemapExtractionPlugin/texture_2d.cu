#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <hip/hip_runtime_api.h>

#define PI 3.1415926536f

// This must be global. Otherwise, hipBindTextureToArray will fail
texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef; 
//texture<float4, 2, hipReadModeElementType> texRef;

__device__ __inline__ float3 make_float3(uchar4& v)
{
	return make_float3(v.x, v.y, v.z);
}

/*
 * Paint a 2D texture with a moving red/green hatch pattern on a
 * strobing blue background.  Note that this kernel reads to and
 * writes from the texture, hence why this texture was not mapped
 * as WriteDiscard.
 */
__global__ void cuda_kernel_texture_2d(uint8_t* buffer, int width, int height, int t)
{
    int x = (blockIdx.x*blockDim.x + threadIdx.x) * 2;
    int y = (blockIdx.y*blockDim.y + threadIdx.y) * 2;

    // in the case where, due to quantization into grids, we have
    // more threads than pixels, skip the threads which don't
    // correspond to valid pixels
    if ((x+1) >= width || (y+1) >= height) return;

	float3 yuv = make_float3(0.0f, 0.0f, 0.0f);

	for (int x_ = x; x_ <= x + 1; x_++)
	{
		for (int y_ = y; y_ <= y + 1; y_++)
		{
			float3 rgb = make_float3(tex2D(texRef, x_, y_));

			yuv.x  =  (0.257f * rgb.z) + (0.504f * rgb.y) + (0.098f * rgb.x) +  16.f;
			yuv.y +=  (0.439f * rgb.z) - (0.368f * rgb.y) - (0.071f * rgb.x) + 128.f;
			yuv.z += -(0.148f * rgb.z) - (0.291f * rgb.y) + (0.439f * rgb.x) + 128.f;
		
			uint8_t* yPtr = (buffer + (height - y_ - 1) * width + x_);
			*yPtr = yuv.x;
		}
	}

	uint8_t* uPtr = (buffer + width * height                        + (height/2 - y/2 - 1) * width/2 + x/2);
	uint8_t* vPtr = (buffer + width * height + (width * height) / 4 + (height/2 - y/2 - 1) * width/2 + x/2);

	*uPtr = yuv.y / 4.f;
	*vPtr = yuv.z / 4.f;
}

extern "C"
void* cuda_texture_2d(hipGraphicsResource* cudaResource, int width, int height, float t)
{
    hipError_t error = hipSuccess;

	//texture<uint8_t, hipTextureType2D, hipReadModeElementType> texRef;
	static int x = 0;
	x++;

	hipArray* cuArray;

	void* cudaLinearMemory;
	//size_t pitch;

	hipMalloc(&cudaLinearMemory, width * height * 4);
	getLastCudaError("hipMallocPitch (g_texture_2d) failed");
	hipMemset(cudaLinearMemory, 0, width * height * 3);

	error = hipGraphicsSubResourceGetMappedArray(&cuArray, cudaResource, 0, 0);
	getLastCudaError("hipGraphicsSubResourceGetMappedArray (cuda_texture_2d) failed");

	
	error = hipBindTextureToArray(texRef, cuArray);
	getLastCudaError("hipGraphicsSubResourceGetMappedArray (cuda_texture_2d) failed");

    dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3(((width/2)+Db.x-1)/Db.x, ((height/2)+Db.y-1)/Db.y);

    cuda_kernel_texture_2d<<<Dg,Db>>>((uint8_t*)cudaLinearMemory, width, height, x);

    error = hipGetLastError();

    if (error != hipSuccess)
    {
        printf("cuda_kernel_texture_2d() failed to launch error = %d\n", error);
    }

	//error = hipMemcpyFromArray(cudaLinearMemory, cuArray, 0, 0, width * height * 4, hipMemcpyDeviceToDevice);

	/*error = hipMemcpy2DToArray(
		cuArray, // dst array
		0, 0,    // offset
		cudaLinearMemory, pitch,       // src
		width * 4 * sizeof(uint8_t), height, // extent
		hipMemcpyDeviceToDevice);*/
	//getLastCudaError("hipMemcpy2DToArray failed");

	return cudaLinearMemory;
}
