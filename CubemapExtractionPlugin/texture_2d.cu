#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <hip/hip_runtime_api.h>

#include "AlloShared/Cubemap.hpp"

// This must be global. Otherwise, hipBindTextureToArray will fail
texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef0;
texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef1;
texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef2;
texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef3;
texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef4;
texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef5;
texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef6;
texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef7;
texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef8;
texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef9;
texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef10;
texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef11;

__device__ __inline__ float3 make_float3(uchar4& v)
{
	return make_float3(v.x, v.y, v.z);
}

__global__ void cuda_kernel_texture_2d(uint8_t* buffer, int width, int height, int face)
{
    int x = (blockIdx.x*blockDim.x + threadIdx.x) * 2;
    int y = (blockIdx.y*blockDim.y + threadIdx.y) * 2;

    // in the case where, due to quantization into grids, we have
    // more threads than pixels, skip the threads which don't
    // correspond to valid pixels
    if ((x+1) >= width || (y+1) >= height) return;

	float3 yuv = make_float3(0.0f, 0.0f, 0.0f);

	for (int x_ = x; x_ <= x + 1; x_++)
	{
		for (int y_ = y; y_ <= y + 1; y_++)
		{
			float3 rgb;
			
			switch (face)
			{
			case 0:
				rgb = make_float3(tex2D(texRef0, x_, y_));
				break;
			case 1:
				rgb = make_float3(tex2D(texRef1, x_, y_));
				break;
			case 2:
				rgb = make_float3(tex2D(texRef2, x_, y_));
				break;
			case 3:
				rgb = make_float3(tex2D(texRef3, x_, y_));
				break;
			case 4:
				rgb = make_float3(tex2D(texRef4, x_, y_));
				break;
			case 5:
				rgb = make_float3(tex2D(texRef5, x_, y_));
				break;
			case 6:
				rgb = make_float3(tex2D(texRef6, x_, y_));
				break;
			case 7:
				rgb = make_float3(tex2D(texRef7, x_, y_));
				break;
			case 8:
				rgb = make_float3(tex2D(texRef8, x_, y_));
				break;
			case 9:
				rgb = make_float3(tex2D(texRef9, x_, y_));
				break;
			case 10:
				rgb = make_float3(tex2D(texRef10, x_, y_));
				break;
			case 11:
				rgb = make_float3(tex2D(texRef11, x_, y_));
				break;
			}
			

			yuv.x  =  (0.257f * rgb.z) + (0.504f * rgb.y) + (0.098f * rgb.x) +  16.f;
			yuv.y +=  (0.439f * rgb.z) - (0.368f * rgb.y) - (0.071f * rgb.x) + 128.f;
			yuv.z += -(0.148f * rgb.z) - (0.291f * rgb.y) + (0.439f * rgb.x) + 128.f;
		
			uint8_t* yPtr = (buffer + (height - y_ - 1) * width + x_);
			*yPtr = yuv.x;
		}
	}

	uint8_t* uPtr = (buffer + width * height                        + (height/2 - y/2 - 1) * width/2 + x/2);
	uint8_t* vPtr = (buffer + width * height + (width * height) / 4 + (height/2 - y/2 - 1) * width/2 + x/2);

	*uPtr = yuv.y / 4.f;
	*vPtr = yuv.z / 4.f;
}

extern "C" void* cuda_texture_2d(hipGraphicsResource* cudaResource, int width, int height, int face)
{
    hipError_t error = hipSuccess;
	hipArray* cuArray;
	void* cudaLinearMemory;

	error = hipMalloc(&cudaLinearMemory, width * height * 4);
	getLastCudaError("hipMallocPitch (g_texture_2d) failed");
	error = hipMemset(cudaLinearMemory, 0, width * height * 3);

	error = hipGraphicsSubResourceGetMappedArray(&cuArray, cudaResource, 0, 0);
	getLastCudaError("hipGraphicsSubResourceGetMappedArray (cuda_texture_2d) failed");

	switch (face)
	{
	case 0:
		error = hipBindTextureToArray(texRef0, cuArray);
		break;
	case 1:
		error = hipBindTextureToArray(texRef1, cuArray);
		break;
	case 2:
		error = hipBindTextureToArray(texRef2, cuArray);
		break;
	case 3:
		error = hipBindTextureToArray(texRef3, cuArray);
		break;
	case 4:
		error = hipBindTextureToArray(texRef4, cuArray);
		break;
	case 5:
		error = hipBindTextureToArray(texRef5, cuArray);
		break;
	case 6:
		error = hipBindTextureToArray(texRef6, cuArray);
		break;
	case 7:
		error = hipBindTextureToArray(texRef7, cuArray);
		break;
	case 8:
		error = hipBindTextureToArray(texRef8, cuArray);
		break;
	case 9:
		error = hipBindTextureToArray(texRef9, cuArray);
		break;
	case 10:
		error = hipBindTextureToArray(texRef10, cuArray);
		break;
	case 11:
		error = hipBindTextureToArray(texRef11, cuArray);
		break;
	}
	
	getLastCudaError("hipGraphicsSubResourceGetMappedArray (cuda_texture_2d) failed");

    dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3(((width/2)+Db.x-1)/Db.x, ((height/2)+Db.y-1)/Db.y);

    cuda_kernel_texture_2d<<<Dg,Db>>>((uint8_t*)cudaLinearMemory, width, height, face);

    error = hipGetLastError();

    if (error != hipSuccess)
    {
        printf("cuda_kernel_texture_2d() failed to launch error = %d\n", error);
    }

	return cudaLinearMemory;
}
