#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <stdint.h>
#include <hip/hip_runtime_api.h>

#define PI 3.1415926536f

// This must be global. Otherwise, hipBindTextureToArray will fail
texture<uchar4, hipTextureType2D, hipReadModeElementType> texRef; 
//texture<float4, 2, hipReadModeElementType> texRef;

/*
 * Paint a 2D texture with a moving red/green hatch pattern on a
 * strobing blue background.  Note that this kernel reads to and
 * writes from the texture, hence why this texture was not mapped
 * as WriteDiscard.
 */
__global__ void cuda_kernel_texture_2d(uint8_t* surface, int width, int height)
{
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;

    // in the case where, due to quantization into grids, we have
    // more threads than pixels, skip the threads which don't
    // correspond to valid pixels
    if (x >= width || y >= height) return;

	uchar4 pixel = tex2D(texRef, x, y);

	uint8_t& yPart = *(surface + y * width + x);

	yPart = 0.299 * pixel.x + 0.587 * pixel.y + 0.114 * pixel.z;
}

extern "C"
void* cuda_texture_2d(hipGraphicsResource* cudaResource, int width, int height, float t)
{
    hipError_t error = hipSuccess;

	//texture<uint8_t, hipTextureType2D, hipReadModeElementType> texRef;

	hipArray* cuArray;

	void* cudaLinearMemory;
	//size_t pitch;

	hipMalloc(&cudaLinearMemory, width * height * 4);
	getLastCudaError("hipMallocPitch (g_texture_2d) failed");
	hipMemset(cudaLinearMemory, 0, width * height * 3);

	error = hipGraphicsSubResourceGetMappedArray(&cuArray, cudaResource, 0, 0);
	getLastCudaError("hipGraphicsSubResourceGetMappedArray (cuda_texture_2d) failed");

	
	error = hipBindTextureToArray(texRef, cuArray);
	getLastCudaError("hipGraphicsSubResourceGetMappedArray (cuda_texture_2d) failed");

    dim3 Db = dim3(16, 16);   // block dimensions are fixed to be 256 threads
    dim3 Dg = dim3((width+Db.x-1)/Db.x, (height+Db.y-1)/Db.y);

    cuda_kernel_texture_2d<<<Dg,Db>>>((uint8_t*)cudaLinearMemory, width, height);

    error = hipGetLastError();

    if (error != hipSuccess)
    {
        printf("cuda_kernel_texture_2d() failed to launch error = %d\n", error);
    }

	//error = hipMemcpyFromArray(cudaLinearMemory, cuArray, 0, 0, width * height * 4, hipMemcpyDeviceToDevice);

	/*error = hipMemcpy2DToArray(
		cuArray, // dst array
		0, 0,    // offset
		cudaLinearMemory, pitch,       // src
		width * 4 * sizeof(uint8_t), height, // extent
		hipMemcpyDeviceToDevice);*/
	//getLastCudaError("hipMemcpy2DToArray failed");

	return cudaLinearMemory;
}
